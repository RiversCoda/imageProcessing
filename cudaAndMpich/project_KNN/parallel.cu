#include "hip/hip_runtime.h"
#include<iostream>  
#include<map>  
#include<vector>  
#include<stdio.h>  
#include<cmath>  
#include<algorithm>  
#include<fstream>  
#include<cstdlib>  
#include<stdlib.h>  
#include "hip/hip_runtime.h"  
#include ""  
using namespace std;  

#define maxCol 20//列数（数据的维度）  
#define testRow 30//测试数据个数
#define maxRow 100
int TDnum = 0;//测试数据的个数统计  
double testData_glo[testRow][maxCol];//存放测试数据集  
string label[testRow];//存放测试数据集标签  
string label1[testRow];//存放测试数据集预测得标签  
  
float gpu_time=0;  
  
__global__ void GPUKNN(double dataset[maxRow][maxCol], double testdata[maxCol], double maps[maxRow])//核函数（训练数据矩阵，测试数据数组，存储距离数组）  
{  
    int i = blockIdx.x * blockDim.x + threadIdx.x;  
    if (i < maxRow)  
        for (int k = 0; k < maxCol; k++)  
            maps[i] += (dataset[i][k] - testdata[k]) * (dataset[i][k] - testdata[k]);//计算欧式距离的平方      
  
}  
  
  
typedef pair<int, double> PAIR;//模板类  
ifstream fin;//输入文件流  
  
class KNN//KNN类  
{  
private:  
    double dataSet[maxRow][maxCol];//训练数据数组  
    string   labels[maxRow];//训练数据的标签数组  
    double testData[maxCol];//一个测试数据数组  
    map<int, double> map_index_dis;//map类模板存储训练数据的索引值和到测试数据的距离，key是数据的编号/序号，value是距离  
    map<string, int> map_label;//map类模板存储训练数据的标签和索引  
public:  
    int k = 5;//初始化KNN算法中的k值大小，k值一般取奇数且一般小于20  
    KNN(char* filename);//有参构造函数  
    void get_all_distance();//获得欧式距离函数  
    string get_max_fre_label();//找到前k个距离最接近标签函数  
    void get_error_rate();//误差率  
    struct CmpByValue//map的pair对按距离大小排序的结构体  
    {  
        bool operator() (const PAIR& lhs, const PAIR& rhs)  
        {  
            return lhs.second < rhs.second;  
        }  
  
    };  
  
};  
KNN::KNN(char* filename)  
{  
    if (TDnum == 0) {  
        fin.open(filename);//打开文件  
        if (!fin)//打开失败  
        {  
            cout << "can not open the file" << endl;  
            exit(0);  
        }  
        for (int i = 0; i < maxRow; i++)//读入训练数据及其标签存储在dataSet和labels中  
        {  
            for (int j = 0; j < maxCol; j++)  
            {  
                fin >> dataSet[i][j];  
  
            }fin >> labels[i];  
  
        }  
        fin.close();//关闭读操作  
    }  
}  
void KNN::get_all_distance()  
{  
    for (int j = 0; j < maxCol; j++)//循环读取测试数据  
    {  
        testData[j] = testData_glo[TDnum][j];  
  
    }  
    double(*gpudS)[maxCol];//dataSet  
    double* gpuTD, * gpuSum;//testData，distance  
  
//记录GPU运行时间  
    hipEvent_t gpu_start, gpu_stop;  
  
    hipEventCreate(&gpu_start);//创建事件对象   
    hipEventCreate(&gpu_stop);  
    hipEventRecord(gpu_start, 0);//记录事件   
  
//分配显存  
    hipMalloc(&gpudS, sizeof(double) * maxCol * maxRow);  
    hipMalloc(&gpuTD, sizeof(double) * maxCol);  
    hipMalloc(&gpuSum, sizeof(double) * maxRow);  
//拷贝  
    hipMemcpy(gpudS, dataSet, sizeof(double) * maxCol * maxRow, hipMemcpyHostToDevice);  
    hipMemcpy(gpuTD, testData, sizeof(double) * maxCol, hipMemcpyHostToDevice);  
    hipMemcpy(gpuSum, 0, sizeof(double) * maxCol, hipMemcpyHostToDevice);//赋值为0  
    double sum[maxRow];//存放拷回的欧式距离的平方  
#ifdef TEST  
    cout << "testdata" << TDnum << ":" << testData[0] << " " << testData[1] << " " << testData[2] << " " << testData[3] << endl;//查看测试数据的各维度值  
#endif // TEST  
      
    GPUKNN << <maxRow, 1 >> > (gpudS, gpuTD, gpuSum);//调用核函数  
  
    hipEventRecord(gpu_stop, 0);  
    hipEventSynchronize(gpu_stop);  
    /* 
    hipEventSynchronize(gpu_stop):使得直到GPU执行完hipEventRecord(gpu_stop, 0)之前的所有语句时， 
    事件gpu_stop才会被记录下来，即起到同步的作用。 
    */  
    float timetemp;  
    hipEventElapsedTime(&timetemp, gpu_start, gpu_stop);//计算两次事件的时间差   
    gpu_time += timetemp;  
  
    hipEventDestroy(gpu_start);//销毁事件对象   
    hipEventDestroy(gpu_stop);  
  
    hipMemcpy(sum, gpuSum, sizeof(double) * maxRow, hipMemcpyDeviceToHost);//拷回距离到sum数组  
    for (int i = 0; i < maxRow; i++)  
    {  
        sum[i] = sqrt(sum[i]);//开方即为欧氏距离  
        map_index_dis[i] = sum[i];//赋值给map
  
    }  
    hipFree(gpudS);//释放  
    hipFree(gpuTD);  
    hipFree(gpuSum);  
}  
  
  
string KNN::get_max_fre_label()  
{  
    vector<pair<int, double> >vec_index_dis(map_index_dis.begin(), map_index_dis.end());  
    sort(vec_index_dis.begin(), vec_index_dis.end(), CmpByValue());//按距离排序  
  
  
    for (int i = 0; i < k; i++)//打印距离最小k个的点的信息  
    {  
#ifdef TEST  
        cout << "index = " << vec_index_dis[i].first << " the distance= " << vec_index_dis[i].second  
            << " the label = " << labels[vec_index_dis[i].first] << endl;  
#endif // TEST  
        map_label[labels[vec_index_dis[i].first]]++;//标签是key，标签数是value  
    }  
  
  
    map<string, int>::iterator itr = map_label.begin();  
    int max_freq = 0;  
    string label;  
    while (itr != map_label.end())//寻找出现次数最多的标签  
    {  
        if (itr->second > max_freq)  
        {  
            max_freq = itr->second;  
            label = itr->first;  
        }  
        itr++;  
    }  
    for (int i = 0; i < k; i++)//恢复初始值0  
    {  
        int xuhao = vec_index_dis[i].first;//距离第i小的点的序号  
        map_label[labels[xuhao]]--;  
    }  
    label1[TDnum] = label;  
    return label;  
}  
void KNN::get_error_rate()  
{  
    int i, count = 0;  
//get testing data and calculate  
    for (i = 0; i < testRow; i++)  
    {  
        if (label[i] != label1[i])  
        {  
            count++;  
        }  
    }  
    cout << "the error rate is = " << (double)count / (double)testRow << endl;  
    cout << "the correct rate is = " << 1.0 - (double)count / (double)testRow << endl;  
}  
  
int main()  
{  
    cout << maxRow << endl;
	cout<<endl<<endl<<endl;
    char* filename = "ring.txt";//训练数据文件名  
    KNN knn(filename);//创建KNN类对象  
    if (TDnum == 0) {//程序开始时导入测试数据集存放至testData_glo数组  
        filename = "ring_test.txt";  
        fin.open(filename);  
        if (!fin)  
        {  
            cout << "can not open the file" << endl;  
            exit(0);  
        }  
        for (int i = 0; i < testRow; i++)  
        {  
            for (int j = 0; j < maxCol; j++)  
            {  
                fin >> testData_glo[i][j];  
  
            }  
            fin >> label[i];  
  
        }  
    }  
    for (int e = 0; e < testRow; e++)//循环完成测试数据的标签预测  
    {  
        knn.get_all_distance();//计算与训练数据的所有距离  
  
        knn.get_max_fre_label();  
        TDnum++;//记录测试数据的个数  
    }  
  
    cout << "k= " << knn.k << " result:" << endl;  
    knn.get_error_rate();  
    cout << "real    label:";  
    for (int i = 0; i < testRow; i++)  
    {  
        cout << label[i]<<" ";  
    }  
    cout << endl; cout << "predict label:";  
    for (int i = 0; i < testRow; i++)  
    {  
        cout << label1[i]<<" ";  
    }  
    cout << endl;  
  
    printf("GPU:Time used:%.3f ms\n", gpu_time);//以毫秒为单位，精度为0.5微秒   
  
    return 0;  
}  


